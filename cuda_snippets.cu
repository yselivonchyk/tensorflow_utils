#include "hip/hip_runtime.h"
// 
// nvcc -arch=sm_70 -o out cuda.cu -run
// !nvprof ./cuda

// Error handling
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
checkCuda( hipDeviceSynchronize() );

errSync  = hipGetLastError();
errAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
if (errSync  != hipSuccess) { printf("Sync  error: %s\n", hipGetErrorString(errSync) ); }
if (errAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(errAsync)); }


// Properties
int deviceId;
hipGetDevice(&deviceId);                 
hipDeviceProp_t props;
hipGetDeviceProperties(&props, deviceId);
computeCapabilityMajor = props.major;
computeCapabilityMinor = props.minor;
multiProcessorCount = props.multiProcessorCount;
warpSize = props.warpSize;

hipDeviceGetAttribute(&smemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, devId);
hipDeviceGetAttribute(&numProcs, hipDeviceAttributeMultiprocessorCount, devId);


//atomic ops
atomicAdd(&a[i], 1);


// memory usage
hipMemPrefetchAsync(a, size, deviceId);
hipMemPrefetchAsync(a, size, hipCpuDeviceId);

//streams
hipStream_t stream;
hipStreamCreate(&stream);
kernel<<<nblocks, nthreads, 0, stream>>>(); 
hipStreamDestroy(stream);
