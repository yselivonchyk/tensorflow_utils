#include "hip/hip_runtime.h"
// 
// nvcc -arch=sm_70 -o out cuda.cu -run
// !nvprof ./cuda

// Error handling
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
checkCuda( hipDeviceSynchronize() );

errSync  = hipGetLastError();
errAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
if (errSync  != hipSuccess) { printf("Sync  error: %s\n", hipGetErrorString(errSync) ); }
if (errAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(errAsync)); }


// Properties
int deviceId;
hipGetDevice(&deviceId);                 
hipDeviceProp_t props;
hipGetDeviceProperties(&props, deviceId);
computeCapabilityMajor = props.major;
computeCapabilityMinor = props.minor;
multiProcessorCount = props.multiProcessorCount;
warpSize = props.warpSize;

hipDeviceGetAttribute(&smemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, devId);
hipDeviceGetAttribute(&numProcs, hipDeviceAttributeMultiprocessorCount, devId);


//atomic ops
atomicAdd(&a[i], 1);


// memory usage
hipMallocManaged(&a, size);
hipMemPrefetchAsync(a, size, deviceId);
hipMemPrefetchAsync(a, size, hipCpuDeviceId);
hipFree(a);

hipMalloc(); // on default GPU
cudaMallocDevice();
hipHostMalloc(); hipHostFree();
hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, steram);


//streams
hipStream_t stream;
hipStreamCreate(&stream);
kernel<<<nblocks, nthreads, 0, stream>>>(); 
hipStreamDestroy(stream); // will keep going until ops are complete
